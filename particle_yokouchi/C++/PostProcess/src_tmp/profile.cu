#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <stdlib.h>

#include "profile.h"
#include "stSetting.h"
#include "stParticle.h"

/* ---------- Class ParticleProfile --------- */
void ParticleProfile::allocate_profile (Setting& setting) {
	// Copy setting
	const	int x_n_rank = setting.X_NUM_RANK;
	const int y_n_rank = setting.Y_NUM_RANK;
	const int x_rank	 = setting.X_RANK;
	const int y_rank	 = setting.Y_RANK;
	const	int	z_domain = setting.Z_DOMAIN;

	// Allocate
	std::cout << "Allocate profile memory -----" << std::endl;
	profile 		= new int[y_n_rank*x_n_rank*z_domain];
	std::cout << "Allocate map memory -----" << std::endl;
	fluid_grid	= new int[y_n_rank*x_n_rank*z_domain];

	// Initilization
	for (int i=0; i<y_n_rank; i++) {
		for (int j=0; j<x_n_rank; j++) {
			for (int k=0; k<z_domain; k++) {
				profile[k+j*z_domain+i*z_domain*x_n_rank]		= 0;
				fluid_grid[k+j*z_domain+i*z_domain*x_n_rank] = x_rank*y_rank;
			}
		}
	}
}

void ParticleProfile::read_map (Setting& setting) {
	// Copy setting
	const int 	x_n_rank	= setting.X_NUM_RANK;
	const int 	y_n_rank	= setting.Y_NUM_RANK;
	const int 	x_rank		= setting.X_RANK;
	const int 	y_rank		= setting.Y_RANK;
	const int 	x_domain 	= setting.X_DOMAIN;
	const int 	y_domain 	= setting.Y_DOMAIN;
	const int 	z_domain	= setting.Z_DOMAIN;
	const float	dx				= setting.dX;
	const char*	fname_map	= setting.FNAME_MAP;

	float	height_map[x_domain*y_domain];
	float max_height = 0.0;
	float sum_height = 0.0;
	int 	count_buil = 0;

	// Read
	std::cout << "Read map. File name >> " << fname_map << std::endl;
	
	std::fstream	mFile;
	mFile.open(fname_map, std::ios::in);
	if(!mFile.is_open()) {
		std::cout << "cannot open map file" << std::endl;
		exit(EXIT_FAILURE);
	}
	int x, y;
	mFile >> x;
	mFile >> y;
	if (x_domain > x || y_domain > y) {
		std::cout << "size of map file is too small" << std::endl;
	}
	for (int i=0; i<y_domain; i++) {
		for (int j=0; j<x_domain; j++) {
			mFile >> height_map[j+i*x_domain];
			sum_height += height_map[j+i*x_domain];
			if (height_map[j+i*x_domain] > 0.0) count_buil++;
			if (max_height < height_map[j+i*x_domain]) max_height = height_map[j+i*x_domain];
		}
	}
	mFile.close();

	// Output map status
	std::cout << " -max height = " << max_height << std::endl;
	std::cout << " -ave height = " << sum_height / count_buil << std::endl;
	std::cout << std::endl;

	// Count fluid grids
	int i, j, ii, jj, k;
	#pragma omp parallel for private(ii,jj,k)
	for (i=0; i<y_n_rank; i++) {
		for (j=0; j<x_n_rank; j++) {
			for (ii=0; ii<y_rank; ii++) {
				for (jj=0; jj<x_rank; jj++) {
					float height 	= height_map[j*jj + i*ii*x_domain];
					int		h_solid	= (int)ceil(height/dx);
					for (k=0; k<h_solid; k++) fluid_grid[k+j*z_domain+i*z_domain*x_n_rank]--;
				}
			}
		}
	}
}

void ParticleProfile::cal_profile (float x, float y, float z, Setting& setting) {
	// Copy setting
	const int x_n_rank 	= setting.X_NUM_RANK;
	const int y_n_rank 	= setting.Y_NUM_RANK;
	const int x_rank	 	= setting.X_RANK;
	const int y_rank	 	= setting.Y_RANK;
	const int z_domain 	= setting.Z_DOMAIN;
	const float dx		 	= setting.dX;
	
	// cal
	if (-0.5<x && x<0.0) x = 0.0;
	if (-0.5<y && y<0.0) y = 0.0;
	if (-0.5<z && z<0.0) z = 0.0;

	const int p_x_rank	= (int)x / (int)((float)x_rank * dx);
	const int p_y_rank	= (int)y / (int)((float)y_rank * dx);
	const	int p_z				= (int)z / (int)dx;

	if (p_x_rank < 0 || x_n_rank-1 < p_x_rank) {
		std::cout << "Error!! (out of domain at profile)\n";
		std::cout << "x = " << x << " : x_rank = " << p_x_rank << std::endl;
	}
	if (p_y_rank < 0 || y_n_rank-1 < p_y_rank) {
		std::cout << "Error!! (out of domain at profile)\n";
		std::cout << "y = " << y << " : y_rank = " << p_y_rank << std::endl;
	}
	if (p_z			 < 0 || z_domain-1 < p_z)	{
		std::cout << "Error!! (out of domain at profile)\n";
		std::cout << "z = " << z << std::endl;
	}

	profile[p_z + p_x_rank*z_domain + p_y_rank*z_domain*x_n_rank]++;
}



void ParticleProfile::output_profile (Setting& setting) {
	// Copy setting
	const int		x_n_rank 	= setting.X_NUM_RANK;
	const int 	y_n_rank 	= setting.Y_NUM_RANK;
	const int 	z_domain 	= setting.Z_DOMAIN;
	const char*	dir_out		= setting.DIR_OUT; 

	// Output
	std::cout << "Output profile >>>>> ";
	std::fstream 	pFile;
	char					profFile[100];

	sprintf(profFile, "./%s/profile.csv", dir_out);
	pFile.open(profFile, std::ios::out);
	for (int i=0; i<y_n_rank; i++) {
		for (int j=0; j<x_n_rank; j++) {
			pFile << "Rank X:: " << j << " : Rank Y:: " << i << std::endl;
			for (int k=0; k<z_domain; k++) {
				pFile << profile[k + j*z_domain + i*z_domain*x_n_rank];
				if (k<z_domain-1) pFile << ",";
			}
			pFile << std::endl;
			for (int k=0; k<z_domain; k++) {
				pFile << fluid_grid[k + j*z_domain + i*z_domain*x_n_rank];
				if (k<z_domain-1) pFile << ",";
			}
			pFile << std::endl;
		}
	}
	pFile.close();
	std::cout << "Finish!!" << std::endl;
}

void ParticleProfile::delete_profile () {
	delete[] profile;
	delete[] fluid_grid;
}
	

