#include <iostream>
#include <time.h>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "residence.h"
#include "stSetting.h"
#include "stParticle.h"

void ParticleResidenceTime::allocate_residence (Setting& setting) {
	// Copy setting 
	const int x_domain = setting.X_DOMAIN;
	const int y_domain = setting.Y_DOMAIN;
	const int n_source = setting.N_SOURCE;

	// Allocate
	residence_particle_h = new float[n_source];
	residence_map      	 = new float[y_domain*x_domain];

	// Allocate device memory
	hipMalloc(&residence_particle_d, n_source*sizeof(float));

	// Inititialize
	for (int i=0; i<n_source; i++) { 
		residence_particle_h[i] = 0.0;
	}

	// Copy memory host -> device
	hipMemcpy(residence_particle_d, residence_particle_h, n_source*sizeof(float), hipMemcpyHostToDevice);	
}

void ParticleResidenceTime::read_source_index (Setting& setting) {
	// Copy setting

}

void ParticleResidenceTime::cal_residence (int t, thrust::host_vector<Particle_Index> p_resid_odd, thrust::host_vector<Particle_Index> p_resid_even,  Setting& setting) {
	// Copy setting
	
}

void ParticleResidenceTime::output_residence (Setting& setting) {

}

void ParticleResidenceTime::delete_residence () {
	delete[] residence_particle_h;
	delete[] residence_map;
	hipFree(residence_particle_d);
}
